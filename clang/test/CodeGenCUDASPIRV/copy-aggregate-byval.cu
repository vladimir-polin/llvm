
#include <hip/hip_runtime.h>
// Tests CUDA kernel arguments get copied by value when targeting SPIR-V, even with
// destructor, copy constructor or move constructor defined by user.

// RUN: %clang -emit-llvm --cuda-device-only --offload=spirv32 \
// RUN:   -nocudalib -nocudainc %s -o %t.bc -c 2>&1
// RUN: llvm-dis -opaque-pointers %t.bc -o %t.ll
// RUN: FileCheck %s --input-file=%t.ll

// RUN: %clang -emit-llvm --cuda-device-only --offload=spirv64 \
// RUN:   -nocudalib -nocudainc %s -o %t.bc -c 2>&1
// RUN: llvm-dis -opaque-pointers %t.bc -o %t.ll
// RUN: FileCheck %s --input-file=%t.ll

class GpuData {
 public:
  __attribute__((host)) __attribute__((device)) GpuData(int* src) {}
  __attribute__((host)) __attribute__((device)) ~GpuData() {}
  __attribute__((host)) __attribute__((device)) GpuData(const GpuData& other) {}
  __attribute__((host)) __attribute__((device)) GpuData(GpuData&& other) {}
};

// CHECK: define
// CHECK-SAME: spir_kernel void @_Z6kernel7GpuData(ptr noundef byval(%class.GpuData) align

__attribute__((global)) void kernel(GpuData output) {}
